

#include <hip/hip_runtime.h>
#include <sys/mman.h>

#include <errno.h>
#include <fcntl.h>
#include <unistd.h>

#include <stdint.h>
#include <stdio.h>

inline
void cuda_assert(hipError_t hipError_t, const char* file, int line, bool abort = true) {
    if(hipError_t == hipSuccess) return;

    fprintf(stderr, "[%s] %s:%d, hipError_t = %d (%s)\n", __FUNCTION__, file, line, hipError_t, hipGetErrorString(hipError_t));
    if(abort) exit(EXIT_FAILURE);
}

#define CUDA_ASSERT(val) { cuda_assert((val), __FILE__, __LINE__); }

__global__
void kernel1(uint32_t* values) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    values[i] = ~i;
}

__host__
int main() {
    CUDA_ASSERT(hipSetDevice(0));
    CUDA_ASSERT(hipSetDeviceFlags(hipDeviceMapHost))

    int fd = open("/dev/dmabuf2", O_RDWR);
    if(fd == -1) {
        printf("F [] open: errno = %d\n", errno);
        return EXIT_FAILURE;
    }

    size_t size = 64 * 1024 * 4096;

    const int nThreadsPerBlock = 1024;
    const int nBlocks = size/4 / nThreadsPerBlock;

    uint32_t* wvalues;
//    wvalues = (uint32_t*)malloc(size);
    hipHostMalloc(&wvalues, size, hipHostMallocDefault);
    for(int i = 0; i < size/4; i++) wvalues[i] = i;
    write(fd, wvalues, size);
    printf("I [] mmap\n");
    wvalues = (uint32_t*)mmap(0, size, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
    if(wvalues == MAP_FAILED) {
        printf("F [] mmap: errno = %d\n", errno);
        return EXIT_FAILURE;
    }
//    if(int error = mlock(wvalues, size)) {
//        printf("F [] mlock: error = %d\n", errno);
//        return EXIT_FAILURE;
//    }
//    CUDA_ASSERT(cudaHostRegister(wvalues, size, cudaHostRegisterPortable));

    // allocate device memory
    uint32_t* values_d;
    printf("I [] cudaMalloc\n");
    hipMalloc(&values_d, size);

    printf("I [] cudaMemcpy\n");
    hipMemcpy(values_d, wvalues, size, hipMemcpyHostToDevice);

    // call kernel
    kernel1<<<nBlocks, nThreadsPerBlock>>>(values_d);

    // allocate host memory
    uint32_t* rvalues;
    rvalues = (uint32_t*)malloc(size);
//    cudaMallocHost(&rvalues, size);

    // copy values from device to host
    hipMemcpy(rvalues, values_d, size, hipMemcpyDeviceToHost);

    // check values
    int error = 0;
    for(int i = 0; i < size/4; i++) {
        if(rvalues[i] == ~wvalues[i]) continue;
        error = 1;
        printf("E [%s] rvalues[%d] = %d\n", __FUNCTION__, i, rvalues[i]);
    }
    if(error == 0) printf("I [%s] OK\n", __FUNCTION__);

    return 0;
}

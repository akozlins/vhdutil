

#include <hip/hip_runtime.h>
#include <cerrno>
#include <cstdint>
#include <cstdio>

#include <fcntl.h>
#include <unistd.h>

#include <sys/mman.h>

inline
void cuda_assert(hipError_t hipError_t, const char* function, const char* file, int line, bool abort = true) {
    if(hipError_t == hipSuccess) return;

    fprintf(stderr, "[%s] %s:%d, hipError_t = %d (%s)\n", function, file, line, hipError_t, hipGetErrorString(hipError_t));
    if(abort) exit(EXIT_FAILURE);
}

#define CUDA_ASSERT(hipError_t) do { cuda_assert((hipError_t), __FUNCTION__, __FILE__, __LINE__); } while(0)

__global__
void kernel1(uint32_t* values) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    values[i] = ~values[i];
}

__host__
int main() {
    int device = 0;
    CUDA_ASSERT(hipSetDevice(device));
    CUDA_ASSERT(hipSetDeviceFlags(hipDeviceMapHost));
    hipDeviceProp_t deviceProperties;
    CUDA_ASSERT(hipGetDeviceProperties(&deviceProperties, device));

    printf("I [] open('/dev/dmabuf0')\n");
    int fd = open("/dev/dmabuf0", O_RDWR);
    if(fd < 0) {
        printf("F [] open: errno = %d\n", errno);
        return EXIT_FAILURE;
    }

    ssize_t size = lseek(fd, 0, SEEK_END);
    printf("I [] size = %ld\n", size);
    if(lseek(fd, 0, SEEK_SET) < 0 || size < 0) {
        printf("F [] lseek < 0\n");
        return EXIT_FAILURE;
    }

    int nThreadsPerBlock = 1;
    while(2 * nThreadsPerBlock <= deviceProperties.maxThreadsPerBlock) nThreadsPerBlock *= 2;
    int nBlocks = size/4 / nThreadsPerBlock;
    printf("I [] nThreadsPerBlock = %d, nBlocks = %d\n", nThreadsPerBlock, nBlocks);

    uint32_t* wvalues;
//    wvalues = (uint32_t*)malloc(size);
//    cudaMallocHost(&wvalues, size);
//    write(fd, wvalues, size);
    printf("I [] mmap\n");
    wvalues = (uint32_t*)mmap(nullptr, size, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
    printf("I [] mmap = %p\n", wvalues);
    for(int i = 0; i < size/4; i++) wvalues[i] = i;
    if(wvalues == MAP_FAILED) {
        printf("F [] mmap: errno = %d\n", errno);
        return EXIT_FAILURE;
    }
//    CUDA_ASSERT(cudaHostRegister(wvalues, size, cudaHostRegisterDefault));

    // allocate device memory
    uint32_t* values_d;
    printf("I [] cudaMalloc\n");
    hipMalloc(&values_d, size);

    printf("I [] cudaMemcpy\n");
    hipMemcpy(values_d, wvalues, size, hipMemcpyHostToDevice);

    // call kernel
    printf("I [] kernel1\n");
    kernel1<<<nBlocks, nThreadsPerBlock>>>(values_d);

    // allocate host memory
    uint32_t* rvalues;
    rvalues = (uint32_t*)malloc(size);
//    cudaMallocHost(&rvalues, size);

    // copy values from device to host
    hipMemcpy(rvalues, values_d, size, hipMemcpyDeviceToHost);

    // check values
    int error = 0;
    for(int i = 0; i < size/4; i++) {
        if(rvalues[i] == ~wvalues[i]) continue;
        error = 1;
        printf("E [%s] rvalues[%d] = %d\n", __FUNCTION__, i, rvalues[i]);
    }
    if(error == 0) printf("I [%s] OK\n", __FUNCTION__);

    return 0;
}
